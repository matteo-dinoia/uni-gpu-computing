#include "hip/hip_runtime.h"
// #define ENABLE_NVTX
// #define ENABLE_CPU_BASELINE
// #define DEBUG_PRINTS
#define ENABLE_CORRECTNESS_CHECK

#define EXIT_INCORRECT_DISTANCES 10

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#ifdef ENABLE_NVTX
#include <nvtx3/nvToolsExt.h>
#endif

#include "../distributed_mmio/include/mmio.h"
#include "../distributed_mmio/include/mmio_utils.h"

#include "../include/colors.h"
#include "../include/utils.cuh"
#include "../include/cli.hpp"
#include "../include/mt19937-64.hpp"
#include "../include/bfs_baseline.cuh"



__global__ void bfs_kernel_push(
    const uint32_t *row_offsets,  // CSR row offsets
    const uint32_t *col_indices,  // CSR column indices (neighbors)
    int *distances,               // Output distances array
    const uint32_t *frontier,     // Current frontier
    uint32_t *next_frontier,      // Next frontier to populate
    const uint32_t frontier_size, // Size of current frontier
    const uint32_t current_level, // BFS level (depth)
    uint32_t *next_frontier_size, // Counter for next frontier
    uint32_t CACHE_SIZE) {
    extern __shared__ uint32_t s_cache[];
    uint32_t cache_used = 0;
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t cache_offset = threadIdx.x * CACHE_SIZE;

    if (tid >= frontier_size) {
        return;
    }

    uint32_t node = frontier[tid];
    uint32_t row_start = row_offsets[node];
    uint32_t row_end = row_offsets[node + 1];

    for (uint32_t i = row_start; i < row_end; i++) {
        uint32_t neighbor = col_indices[i];

        // Use atomic compare-and-swap to avoid revisiting nodes
        if (atomicCAS(&distances[neighbor], -1, current_level + 1) == -1) {
            // Atomically add the neighbor to the next frontier
            s_cache[cache_offset + cache_used++] = neighbor;
            if (cache_used >= CACHE_SIZE) {
                uint32_t offset = atomicAdd(next_frontier_size, cache_used);
                memcpy(&next_frontier[offset], &s_cache[cache_offset], sizeof(uint32_t) * cache_used);
                cache_used = 0;
            }
        }
    }

    if (cache_used > 0) {
        uint32_t offset = atomicAdd(next_frontier_size, cache_used);
        memcpy(&next_frontier[offset], &s_cache[cache_offset], sizeof(uint32_t) * cache_used);
        cache_used = 0;
    }
}


void gpu_bfs(
    const uint32_t N,
    const uint32_t M,
    const uint32_t *h_rowptr,
    const uint32_t *h_colidx,
    const uint32_t source,
    int *h_distances) {
    float tot_time = 0.0;
    CUDA_TIMER_INIT(H2D_copy)

    // Allocate and copy graph to device
    uint32_t *d_row_offsets;
    uint32_t *d_col_indices;
    CHECK_CUDA(hipMalloc(&d_row_offsets, (N + 1) * sizeof(uint32_t)));
    CHECK_CUDA(hipMalloc(&d_col_indices, M * sizeof(uint32_t)));
    CHECK_CUDA(hipMemcpy(d_row_offsets, h_rowptr, (N + 1) * sizeof(uint32_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_col_indices, h_colidx, M * sizeof(uint32_t), hipMemcpyHostToDevice));

    // Allocate memory for distances and frontier queues
    int *d_distances;
    uint32_t *d_frontier;
    uint32_t *d_next_frontier;
    uint32_t *d_next_frontier_size;
    CHECK_CUDA(hipMalloc(&d_distances, N * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_frontier, N * sizeof(uint32_t)));
    CHECK_CUDA(hipMalloc(&d_next_frontier, N * sizeof(uint32_t)));
    CHECK_CUDA(hipMalloc(&d_next_frontier_size, sizeof(uint32_t)));

    std::vector<uint32_t> h_frontier(N);
    h_frontier[0] = source;

    CHECK_CUDA(hipMemcpy(d_frontier, h_frontier.data(), sizeof(uint32_t), hipMemcpyHostToDevice));
    // Initialize all distances to -1 (unvisited), and source distance to 0
    CHECK_CUDA(hipMemset(d_distances, -1, N * sizeof(int)));
    CHECK_CUDA(hipMemset(d_distances + source, 0, sizeof(int))); // set to 0

    CUDA_TIMER_STOP(H2D_copy)
#ifdef DEBUG_PRINTS
    CUDA_TIMER_PRINT(H2D_copy)
#endif
    tot_time += CUDA_TIMER_ELAPSED(H2D_copy);
    CUDA_TIMER_DESTROY(H2D_copy)

    uint32_t current_frontier_size = 1;
    uint32_t level = 0;


    uint32_t cache_single_size = 32;
    //    if (M >= 100e6) {
    //        cache_single_size = 0;
    //    } else if (N < 2.5e6) {
    //        cache_single_size = 8;
    //    } else if (N < 8e6) {
    //        cache_single_size = 16;
    //    } else if (N < 30e6) {
    //        cache_single_size = 32;
    //    } else {
    //        cache_single_size = 1;
    //    }


    // Main BFS loop
    CPU_TIMER_INIT(BASELINE_BFS)
    while (current_frontier_size > 0) {

#ifdef DEBUG_PRINTS
        printf("[GPU BFS%s] level=%u, current_frontier_size=%u\n", is_placeholder ? "" : " BASELINE", level, current_frontier_size);
#endif
#ifdef ENABLE_NVTX
        // Mark start of level in NVTX
        nvtxRangePushA(("BFS Level " + std::to_string(level)).c_str());
#endif

        // Reset counter for next frontier
        CHECK_CUDA(hipMemset(d_next_frontier_size, 0, sizeof(uint32_t)));

        uint32_t block_size = 32;
        //        uint32_t num_blocks = 256;
        uint32_t num_blocks = CEILING(current_frontier_size, block_size);

        if (cache_single_size > 0) {
            uint32_t cache_size = block_size * (cache_single_size * sizeof(uint32_t));


            // CUDA_TIMER_INIT(BFS_kernel)
            //        printf("%u %u %u\n", num_blocks, block_size, cache_size);
            bfs_kernel_push<<<num_blocks, block_size, cache_size>>>(
                d_row_offsets,
                d_col_indices,
                d_distances,
                d_frontier,
                d_next_frontier,
                current_frontier_size,
                level,
                d_next_frontier_size,
                cache_single_size);
        } else {
            bfs_kernel_baseline<<<num_blocks, block_size>>>(
                d_row_offsets,
                d_col_indices,
                d_distances,
                d_frontier,
                d_next_frontier,
                current_frontier_size,
                level,
                d_next_frontier_size);
        }

        CHECK_CUDA(hipDeviceSynchronize());

        // Swap frontier pointers
        std::swap(d_frontier, d_next_frontier);

        // Copy size of next frontier to host
        CHECK_CUDA(hipMemcpy(&current_frontier_size, d_next_frontier_size, sizeof(uint32_t), hipMemcpyDeviceToHost));
        level++;

#ifdef ENABLE_NVTX
        // End NVTX range for level
        nvtxRangePop();
#endif
    }
    CPU_TIMER_STOP(BASELINE_BFS)
#ifdef DEBUG_PRINTS
    CPU_TIMER_PRINT(BASELINE_BFS)
#endif
    tot_time += CPU_TIMER_ELAPSED(BASELINE_BFS);

    CUDA_TIMER_INIT(D2H_copy)
    CHECK_CUDA(hipMemcpy(h_distances, d_distances, N * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_TIMER_STOP(D2H_copy)
#ifdef DEBUG_PRINTS
    CUDA_TIMER_PRINT(D2H_copy)
#endif
    tot_time += CUDA_TIMER_ELAPSED(D2H_copy);
    CUDA_TIMER_DESTROY(D2H_copy);

    printf("\n[OUT] Total BFS time: %f ms\n" RESET, tot_time);

    // Free device memory
    hipFree(d_row_offsets);
    hipFree(d_col_indices);
    hipFree(d_distances);
    hipFree(d_frontier);
    hipFree(d_next_frontier);
    hipFree(d_next_frontier_size);
}


int main(int argc, char **argv) {
    int return_code = EXIT_SUCCESS;

    Cli_Args args;
    init_cli();
    if (parse_args(argc, argv, &args) != 0) {
        return -1;
    }

    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count <= 0) {
        fprintf(stderr, "No GPU available: device_count=%d\n", device_count);
        return EXIT_FAILURE;
    }
    hipSetDevice(0);

    CPU_TIMER_INIT(MTX_read)
    CSR_local<uint32_t, float> *csr = Distr_MMIO_CSR_local_read<uint32_t, float>(args.filename);
    if (csr == NULL) {
        printf("Failed to import graph from file [%s]\n", args.filename);
        return -1;
    }
    CPU_TIMER_STOP(MTX_read)
    printf("\n[OUT] MTX file read time: %f ms\n", CPU_TIMER_ELAPSED(MTX_read));
    printf("Graph size: %.3fM vertices, %.3fM edges\n", csr->nrows / 1e6, csr->nnz / 1e6);

    GraphCSR graph;
    graph.row_ptr = csr->row_ptr;
    graph.col_idx = csr->col_idx;
    graph.num_vertices = csr->nrows;
    graph.num_edges = csr->nnz;
    // print_graph_csr(graph);

    uint32_t *sources = generate_sources(&graph, args.runs, graph.num_vertices, args.source);
    int *distances_gpu_baseline = (int *)malloc(graph.num_vertices * sizeof(int));
    int *distances = (int *)malloc(graph.num_vertices * sizeof(int));
    bool correct = true;

    for (int source_i = 0; source_i < args.runs; source_i++) {
        uint32_t source = sources[source_i];
        printf("\n[OUT] -- BFS iteration #%u, source=%u --\n", source_i, source);

        // Run the BFS baseline
        gpu_bfs_baseline(graph.num_vertices, graph.num_edges, graph.row_ptr, graph.col_idx, source, distances_gpu_baseline, false);

#ifdef ENABLE_NVTX
        nvtxRangePushA("Complete BFS");
#endif
        gpu_bfs(graph.num_vertices, graph.num_edges, graph.row_ptr, graph.col_idx, source, distances);
#ifdef ENABLE_NVTX
        nvtxRangePop();
#endif

        bool match = true;
#ifdef ENABLE_CORRECTNESS_CHECK
        for (uint32_t i = 0; i < graph.num_vertices; ++i) {
            if (distances_gpu_baseline[i] != distances[i]) {
                printf("Mismatch at node %u: Baseline distance = %d, Your distance = %d\n", i, distances_gpu_baseline[i], distances[i]);
                match = false;
                break;
            }
        }
        if (match) {
            printf(BRIGHT_GREEN "Correctness OK\n" RESET);
        } else {
            printf(BRIGHT_RED "GPU and CPU BFS results do not match for source node %u.\n" RESET, source);
            return_code = EXIT_INCORRECT_DISTANCES;
            correct = false;
        }
#endif

#ifdef ENABLE_CPU_BASELINE
        int cpu_distances[graph.num_vertices];

        CPU_TIMER_INIT(CPU_BFS)
        cpu_bfs_baseline(graph.num_vertices, graph.row_ptr, graph.col_idx, source, cpu_distances);
        CPU_TIMER_CLOSE(CPU_BFS)

        match = true;
        for (uint32_t i = 0; i < graph.num_vertices; ++i) {
            if (distances_gpu_baseline[i] != cpu_distances[i]) {
                printf("Mismatch at node %u: GPU distance = %d, CPU distance = %d\n", i, distances_gpu_baseline[i], cpu_distances[i]);
                match = false;
                break;
            }
        }
        if (match) {
            printf(BRIGHT_GREEN "[CPU] Correctness OK\n" RESET);
        } else {
            printf(BRIGHT_RED "GPU and CPU BFS results do not match for source node %u.\n" RESET, source);
            return_code = EXIT_INCORRECT_DISTANCES;
        }
#endif
    }

    if (correct)
        printf("\n[OUT] ALL RESULTS ARE CORRECT\n");
    else
        printf(BRIGHT_RED "\nSOME RESULTS ARE WRONG\n" RESET);

    Distr_MMIO_CSR_local_destroy(&csr);
    free(sources);
    free(distances_gpu_baseline);
    free(distances);

    return return_code;
}
