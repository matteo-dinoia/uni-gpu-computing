#include "hip/hip_runtime.h"
// #define ENABLE_NVTX
// #define ENABLE_CPU_BASELINE
// #define DEBUG_PRINTS
#define ENABLE_CORRECTNESS_CHECK

#define EXIT_INCORRECT_DISTANCES 10

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#ifdef ENABLE_NVTX
#include <nvtx3/nvToolsExt.h>
#endif

#include "../distributed_mmio/include/mmio.h"
#include "../distributed_mmio/include/mmio_utils.h"

#include "../include/colors.h"
#include "../include/utils.cuh"
#include "../include/cli.hpp"
#include "../include/mt19937-64.hpp"
#include "../include/bfs_baseline.cuh"

// Kernel: Process each node in the frontier and add unvisited neighbors to next_frontier
/*__global__ void bfs_kernel_pull(
    const uint32_t N,             // Number of nodes
    const uint32_t *row_offsets,  // CSR row offsets
    const uint32_t *col_indices,  // CSR column indices (neighbors)
    int *distances,               // Output distances array
    const uint32_t *frontier,     // Current frontier
    uint32_t *next_frontier,      // Next frontier to populate
    const uint32_t frontier_size, // Size of current frontier
    const uint32_t current_level, // BFS level (depth)
    uint32_t *next_frontier_size  // Counter for next frontier
    ) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t n_threads = gridDim.x * blockDim.x;

    for (uint32_t node = tid; node < N; node += n_threads) {
        if (distances[node] != -1) {
            continue;
        }

        uint32_t row_start = row_offsets[node];
        uint32_t row_end = row_offsets[node + 1];

        for (uint32_t i = row_start; i < row_end; i++) {
            uint32_t neighbor = col_indices[i];

            if (distances[neighbor] == current_level) {
                distances[node] = current_level + 1;
                // Atomically add the neighbor to the next frontier
                uint32_t index = atomicAdd(next_frontier_size, 1);
                next_frontier[index] = node;
                break;
            }
        }
    }
}*/

__global__ void bfs_kernel_push(
    const uint32_t *row_offsets,  // CSR row offsets
    const uint32_t *col_indices,  // CSR column indices (neighbors)
    int *distances,               // Output distances array
    const uint32_t *frontier,     // Current frontier
    uint32_t *next_frontier,      // Next frontier to populate
    const uint32_t frontier_size, // Size of current frontier
    const uint32_t current_level, // BFS level (depth)
    uint32_t *next_frontier_size  // Counter for next frontier
    ) {

    const uint32_t n_threads = gridDim.x * blockDim.x;
    const uint32_t per_thread = (int)ceil(frontier_size / (float)n_threads);
    const uint32_t start_i = blockIdx.x * blockDim.x + threadIdx.x;

    const uint32_t wrap_id = start_i / warpSize;
    const uint32_t start = wrap_id * (per_thread * warpSize) + (start_i - wrap_id * warpSize);
    const uint32_t incr = warpSize;

    for (int i = 0; i < per_thread; i++) {
        const int el = start + i * incr;
        if (el < frontier_size) {
            uint32_t node = frontier[el];
            uint32_t row_start = row_offsets[node];
            uint32_t row_end = row_offsets[node + 1];

            for (uint32_t i = row_start; i < row_end; i++) {
                uint32_t neighbor = col_indices[i];

                // Use atomic compare-and-swap to avoid revisiting nodes
                if (atomicCAS(&distances[neighbor], -1, current_level + 1) == -1) {
                    // Atomically add the neighbor to the next frontier
                    uint32_t index = atomicAdd(next_frontier_size, 1);
                    next_frontier[index] = neighbor;
                }
            }
        }
    }
}



void gpu_bfs(
    const uint32_t N,
    const uint32_t M,
    const uint32_t *h_rowptr,
    const uint32_t *h_colidx,
    const uint32_t source,
    int *h_distances) {
    float tot_time = 0.0;
    CUDA_TIMER_INIT(H2D_copy)

    // Allocate and copy graph to device
    uint32_t *d_row_offsets;
    uint32_t *d_col_indices;
    CHECK_CUDA(hipMalloc(&d_row_offsets, (N + 1) * sizeof(uint32_t)));
    CHECK_CUDA(hipMalloc(&d_col_indices, M * sizeof(uint32_t)));
    CHECK_CUDA(hipMemcpy(d_row_offsets, h_rowptr, (N + 1) * sizeof(uint32_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_col_indices, h_colidx, M * sizeof(uint32_t), hipMemcpyHostToDevice));

    // Allocate memory for distances and frontier queues
    int *d_distances;
    uint32_t *d_frontier;
    uint32_t *d_next_frontier;
    uint32_t *d_next_frontier_size;

    CHECK_CUDA(hipMalloc(&d_distances, N * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_frontier, N * sizeof(uint32_t)));
    CHECK_CUDA(hipMalloc(&d_next_frontier, N * sizeof(uint32_t)));
    CHECK_CUDA(hipMalloc(&d_next_frontier_size, sizeof(uint32_t)));

    std::vector<uint32_t> h_frontier(N);
    h_frontier[0] = source;

    CHECK_CUDA(hipMemcpy(d_frontier, h_frontier.data(), sizeof(uint32_t), hipMemcpyHostToDevice));
    // Initialize all distances to -1 (unvisited), and source distance to 0
    CHECK_CUDA(hipMemset(d_distances, -1, N * sizeof(int)));
    CHECK_CUDA(hipMemset(d_distances + source, 0, sizeof(int))); // set to 0

    CUDA_TIMER_STOP(H2D_copy)
#ifdef DEBUG_PRINTS
    CUDA_TIMER_PRINT(H2D_copy)
#endif
    tot_time += CUDA_TIMER_ELAPSED(H2D_copy);
    CUDA_TIMER_DESTROY(H2D_copy)

    uint32_t current_frontier_size = 1;
    uint32_t level = 0;

    //uint32_t curr_edges = 0;
    //uint32_t *frontier = (uint32_t *)malloc(N * sizeof(uint32_t));
    //frontier[0] = source;
    //uint32_t entropy = 0;
    //bool push_mode = true;
    // uint32_t unvisited_edges = M;

    // Main BFS loop
    CPU_TIMER_INIT(BASELINE_BFS)
    while (current_frontier_size > 0) {

#ifdef DEBUG_PRINTS
        printf("[GPU BFS%s] level=%u, current_frontier_size=%u\n", is_placeholder ? "" : " BASELINE", level, current_frontier_size);
#endif
#ifdef ENABLE_NVTX
        // Mark start of level in NVTX
        nvtxRangePushA(("BFS Level " + std::to_string(level)).c_str());
#endif

        // Reset counter for next frontier
        CHECK_CUDA(hipMemset(d_next_frontier_size, 0, sizeof(uint32_t)));
        CHECK_CUDA(hipMemset(d_next_frontier_size, 0, sizeof(uint32_t)));



        /*uint32_t edge_from_frontier = 0;
        for (uint32_t i = 0; i < current_frontier_size; i++) {
            uint32_t node = frontier[i];
            edge_from_frontier += h_rowptr[node + 1] - h_rowptr[node];
        }
        edge_from_frontier -= curr_edges;
        printf("%u %u %u %u\n", edge_from_frontier, curr_edges, unvisited_edges, current_frontier_size);

        if (push_mode && edge_from_frontier > unvisited_edges / 5) {
            push_mode = !push_mode;
        } else if (!push_mode && current_frontier_size < N / 24) {
            push_mode = !push_mode;
        }
        //printf("%d", push_mode ? 1 : 0);
        uint32_t block_size = 512;

        entropy += push_mode ? 1 : 0;
        if (true) {
            uint32_t num_blocks = CEILING(current_frontier_size, block_size);
            bfs_kernel_push<<<num_blocks, block_size>>>(
                d_row_offsets,
                d_col_indices,
                d_distances,
                d_frontier,
                d_next_frontier,
                current_frontier_size,
                level,
                d_next_frontier_size);
        } else {
            uint32_t num_blocks = std::min(CEILING(N, block_size), 256u);
            bfs_kernel_pull<<<num_blocks, block_size>>>(
                N,
                d_row_offsets,
                d_col_indices,
                d_distances,
                d_frontier,
                d_next_frontier,
                current_frontier_size,
                level,
                d_next_frontier_size);
        }*/
        uint32_t block_size = 512;
        uint32_t num_blocks = std::min(CEILING(current_frontier_size, block_size), 256u);
        bfs_kernel_push<<<num_blocks, block_size>>>(
            d_row_offsets,
            d_col_indices,
            d_distances,
            d_frontier,
            d_next_frontier,
            current_frontier_size,
            level,
            d_next_frontier_size);


        CHECK_CUDA(hipDeviceSynchronize());
        // CUDA_TIMER_STOP(BFS_kernel)
        // #ifdef DEBUG_PRINTS
        //   CUDA_TIMER_PRINT(BFS_kernel)
        // #endif
        // CUDA_TIMER_DESTROY(BFS_kernel)

        // Swap frontier pointers
        std::swap(d_frontier, d_next_frontier);

        // Copy size of next frontier to host
        CHECK_CUDA(hipMemcpy(&current_frontier_size, d_next_frontier_size, sizeof(uint32_t), hipMemcpyDeviceToHost));
        //CHECK_CUDA(hipMemcpy(frontier, d_frontier, sizeof(uint32_t) * current_frontier_size, hipMemcpyDeviceToHost));

        //CHECK_CUDA(hipMemcpy(&curr_edges, d_curr_edges, sizeof(uint32_t), hipMemcpyDeviceToHost));
        //curr_edges = edge_from_frontier;
        //unvisited_edges -= curr_edges;
        level++;

#ifdef ENABLE_NVTX
        // End NVTX range for level
        nvtxRangePop();
#endif
    }
    CPU_TIMER_STOP(BASELINE_BFS)
#ifdef DEBUG_PRINTS
    CPU_TIMER_PRINT(BASELINE_BFS)
#endif
    tot_time += CPU_TIMER_ELAPSED(BASELINE_BFS);

    CUDA_TIMER_INIT(D2H_copy)
    CHECK_CUDA(hipMemcpy(h_distances, d_distances, N * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_TIMER_STOP(D2H_copy)
#ifdef DEBUG_PRINTS
    CUDA_TIMER_PRINT(D2H_copy)
#endif
    tot_time += CUDA_TIMER_ELAPSED(D2H_copy);
    CUDA_TIMER_DESTROY(D2H_copy)

    // This output format is MANDATORY, DO NOT CHANGE IT
    //printf("\n%d/%d\n", entropy, level);
    printf("\n[OUT] Total BFS time: %f ms\n" RESET, tot_time);

    // Free device memory
    hipFree(d_row_offsets);
    hipFree(d_col_indices);
    hipFree(d_distances);
    hipFree(d_frontier);
    hipFree(d_next_frontier);
    hipFree(d_next_frontier_size);
}

void gpu_bfs_bkp(
    const uint32_t N,         // Number of veritices
    const uint32_t M,         // Number of edges
    const uint32_t *h_rowptr, // Graph CSR rowptr
    const uint32_t *h_colidx, // Graph CSR colidx
    const uint32_t source,    // Source veritex
    int *h_distances          // Write here your distances
    ) {
    /***********************
   * IMPLEMENT HERE YOUR CUDA BFS
   * Feel free to structure you code (i.e. create other files, macros etc.)
   * *********************/


    // !! This is an example of how to keep track of runtime. Make sure to include everything. !!
    float tot_time = 0.0f;
    CPU_TIMER_INIT(BFS_preprocess)

    //<<< preprocess >>>

    CHECK_CUDA(hipDeviceSynchronize());
    CPU_TIMER_STOP(BFS_preprocess)
    tot_time += CPU_TIMER_ELAPSED(BFS_preprocess);
    CPU_TIMER_PRINT(BFS_preprocess)

    CPU_TIMER_INIT(BFS)

    //<<< kernel >>>
    gpu_bfs_baseline(N, M, h_rowptr, h_colidx, source, h_distances, true);

    CHECK_CUDA(hipDeviceSynchronize());
    CPU_TIMER_STOP(BFS)
    tot_time += CPU_TIMER_ELAPSED(BFS);
    CPU_TIMER_PRINT(BFS)
    CPU_TIMER_INIT(BFS_postprocess)

    //<<< postprocess >>>

    CHECK_CUDA(hipDeviceSynchronize());
    CPU_TIMER_STOP(BFS_postprocess)
    tot_time += CPU_TIMER_ELAPSED(BFS_postprocess);
    CPU_TIMER_PRINT(BFS_postprocess)

    // This output format is MANDATORY, DO NOT CHANGE IT
    printf("\n[OUT] Total BFS time: %f ms\n" RESET, tot_time);
}

int main(int argc, char **argv) {
    int return_code = EXIT_SUCCESS;

    Cli_Args args;
    init_cli();
    if (parse_args(argc, argv, &args) != 0) {
        return -1;
    }

    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count <= 0) {
        fprintf(stderr, "No GPU available: device_count=%d\n", device_count);
        return EXIT_FAILURE;
    }
    hipSetDevice(0);

    CPU_TIMER_INIT(MTX_read)
    CSR_local<uint32_t, float> *csr = Distr_MMIO_CSR_local_read<uint32_t, float>(args.filename);
    if (csr == NULL) {
        printf("Failed to import graph from file [%s]\n", args.filename);
        return -1;
    }
    CPU_TIMER_STOP(MTX_read)
    printf("\n[OUT] MTX file read time: %f ms\n", CPU_TIMER_ELAPSED(MTX_read));
    printf("Graph size: %.3fM vertices, %.3fM edges\n", csr->nrows / 1e6, csr->nnz / 1e6);

    GraphCSR graph;
    graph.row_ptr = csr->row_ptr;
    graph.col_idx = csr->col_idx;
    graph.num_vertices = csr->nrows;
    graph.num_edges = csr->nnz;
    // print_graph_csr(graph);

    uint32_t *sources = generate_sources(&graph, args.runs, graph.num_vertices, args.source);
    int *distances_gpu_baseline = (int *)malloc(graph.num_vertices * sizeof(int));
    int *distances = (int *)malloc(graph.num_vertices * sizeof(int));
    bool correct = true;

    for (int source_i = 0; source_i < args.runs; source_i++) {
        uint32_t source = sources[source_i];
        printf("\n[OUT] -- BFS iteration #%u, source=%u --\n", source_i, source);

        // Run the BFS baseline
        gpu_bfs_baseline(graph.num_vertices, graph.num_edges, graph.row_ptr, graph.col_idx, source, distances_gpu_baseline, false);

#ifdef ENABLE_NVTX
        nvtxRangePushA("Complete BFS");
#endif
        gpu_bfs(graph.num_vertices, graph.num_edges, graph.row_ptr, graph.col_idx, source, distances);
#ifdef ENABLE_NVTX
        nvtxRangePop();
#endif

        bool match = true;
#ifdef ENABLE_CORRECTNESS_CHECK
        for (uint32_t i = 0; i < graph.num_vertices; ++i) {
            if (distances_gpu_baseline[i] != distances[i]) {
                printf("Mismatch at node %u: Baseline distance = %d, Your distance = %d\n", i, distances_gpu_baseline[i], distances[i]);
                match = false;
                break;
            }
        }
        if (match) {
            printf(BRIGHT_GREEN "Correctness OK\n" RESET);
        } else {
            printf(BRIGHT_RED "GPU and CPU BFS results do not match for source node %u.\n" RESET, source);
            return_code = EXIT_INCORRECT_DISTANCES;
            correct = false;
        }
#endif

#ifdef ENABLE_CPU_BASELINE
        int cpu_distances[graph.num_vertices];

        CPU_TIMER_INIT(CPU_BFS)
        cpu_bfs_baseline(graph.num_vertices, graph.row_ptr, graph.col_idx, source, cpu_distances);
        CPU_TIMER_CLOSE(CPU_BFS)

        match = true;
        for (uint32_t i = 0; i < graph.num_vertices; ++i) {
            if (distances_gpu_baseline[i] != cpu_distances[i]) {
                printf("Mismatch at node %u: GPU distance = %d, CPU distance = %d\n", i, distances_gpu_baseline[i], cpu_distances[i]);
                match = false;
                break;
            }
        }
        if (match) {
            printf(BRIGHT_GREEN "[CPU] Correctness OK\n" RESET);
        } else {
            printf(BRIGHT_RED "GPU and CPU BFS results do not match for source node %u.\n" RESET, source);
            return_code = EXIT_INCORRECT_DISTANCES;
        }
#endif
    }

    if (correct)
        printf("\n[OUT] ALL RESULTS ARE CORRECT\n");
    else
        printf(BRIGHT_RED "\nSOME RESULTS ARE WRONG\n" RESET);

    Distr_MMIO_CSR_local_destroy(&csr);
    free(sources);
    free(distances_gpu_baseline);
    free(distances);

    return return_code;
}
